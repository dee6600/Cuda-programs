#include<hip/hip_runtime.h>

int main()
{
    hipDeviceProp_t p;
    int device_id;
    int major;
    int minor;


    hipGetDevice(&device_id);
    hipGetDeviceProperties(&p,device_id);

    major=p.major;
    minor=p.minor;

    printf("Name of GPU on your system is %s\n",p.name);

    printf("\n Compute Capability of a current GPU on your system is %d.%d",major,minor);

    return 0;
}

