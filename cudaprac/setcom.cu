#include<stdio.h>
#include<hip/hip_runtime.h>

int main()
{
hipDeviceProp_t p;
int count,i,flag=0;

hipGetDeviceCount(&count);

for(i=0;i<count;i++)
{
    hipGetDeviceProperties(&p,i);


        if(p.major==1 && p.minor==2)
        {
            hipSetDevice(i);
            printf("GPU with Compute Capability 1.2 is set as current GPU on your system.\n");
            flag=1;
        }




